#include "hip/hip_runtime.h"
#include "matrixmul_kernel.h"

#define BLOCK_SIZE 16

__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P) {
	int wM = M.width;
	int wN = N.width;

	const unsigned int bx = blockIdx.x;
	const unsigned int by = blockIdx.y;
	const unsigned int tx = threadIdx.x;
	const unsigned int ty = threadIdx.y;

	const unsigned int mBegin = wM * (by * BLOCK_SIZE);
	const unsigned int mEnd = mBegin + wM;
	const unsigned int mStep = BLOCK_SIZE;

	const unsigned int nBegin = BLOCK_SIZE * bx;
	const unsigned int nStep = BLOCK_SIZE * wN;

	float Psub = 0;

	unsigned int m, n;

	// Perform all non-edge case multiplications
	for (m = mBegin, n = nBegin; m < mEnd - BLOCK_SIZE; m += mStep, n += nStep) {
		__shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

		Ms[ty][tx] = M.elements[m + wM * ty + tx];
		Ns[ty][tx] = N.elements[n + wN * ty + tx];

		__syncthreads();

		for (int k = 0; k < BLOCK_SIZE; k++) {
			Psub += Ms[ty][k] * Ns[k][tx];
		}

		__syncthreads();
	}

	// Perform edge case multiplications
	__shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

	bool c1 = ((m - mBegin) + tx < wM);
	bool c2 = (ty + (m - mBegin) < wM);
	bool c3 = (bx * BLOCK_SIZE + tx) < wN;

	if (c1)
		Ms[ty][tx] = M.elements[m + wM * ty + tx];
	else
		Ms[ty][tx] = 0;
	if (c2 && c3)
		Ns[ty][tx] = N.elements[n + wN * ty + tx];
	else
		Ns[ty][tx] = 0;

	__syncthreads();

	for (int k = 0; k < BLOCK_SIZE; k++) {
		Psub += Ms[ty][k] * Ns[k][tx];
	}
	__syncthreads();

	if (c3) {
        int p = wN * BLOCK_SIZE * by + BLOCK_SIZE * bx;
        P.elements[p + wN * ty + tx] = Psub;
    }
}
